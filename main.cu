#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
	{
		C[i] = A[i] + B[i];
	}
}

void vecAdd(float* A, float* B, float* C, int n)
{
	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_B, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_C, size);

	vecAddKernel <<< ceil(n / 256.0), 256 >>> (d_A, d_B, d_C, n);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

int main()
{
	float A[5] = { 1, 2, 3, 4, 5 };
	float B[5] = { 10, 20, 30, 40, 50 };
	float C[5];

	cout << "A:";
	for (int i = 0; i < 5; i++)
	{
		cout << A[i] << " ";
	}
	cout << endl;

	cout << "B:";
	for (int i = 0; i < 5; i++)
	{
		cout << B[i] << " ";
	}
	cout << endl;

	vecAdd(A, B, C, 5);

	cout << "C:";
	for (int i = 0; i < 5; i++)
	{
		cout << C[i] << " ";
	}
	cout << endl;


	// cout << "Hello" << endl;
}